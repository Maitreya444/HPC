#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    
    if (deviceCount == 0) {
        std::cout << "No CUDA devices available" << std::endl;
    } else {
        std::cout << "CUDA devices available: " << deviceCount << std::endl;
    }
    
    return 0;
}
