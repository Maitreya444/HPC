#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 4;

__global__ void add(int* A, int* B, int* C, int size) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) 
    {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    int A[N] = {3, 6, 7, 5};
    int B[N] = {3, 5, 6, 2};
    int C[N] = {0};

    int *d_A, *d_B, *d_C;
    int size = N * sizeof(int);

    // Allocate memory on device
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch kernel
    add<<<1, N>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    std::cout << "Vector A: ";
    for (int i = 0; i < N; ++i) {
        std::cout << A[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Vector B: ";
    for (int i = 0; i < N; ++i) {
        std::cout << B[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Addition: ";
    for (int i = 0; i < N; ++i) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
